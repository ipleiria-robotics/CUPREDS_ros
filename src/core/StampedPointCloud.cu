/*
* Copyright (c) 2023 Carlos Tojal.
* All rights reserved.
*
* Reference a pointcloud and its capture timestamp on the same class.
* Useful for point cloud management.
*/

#include "StampedPointCloud.h"

StampedPointCloud::StampedPointCloud(std::string originTopic) {
    this->timestamp = Utils::getCurrentTimeMillis();

    this->setOriginTopic(originTopic);

    this->label = generateLabel();

    this->cloud = pcl::PointCloud<pcl::PointXYZRGBL>().makeShared();
}

std::uint32_t StampedPointCloud::generateLabel() {

    std::string combined = this->originTopic + std::to_string(this->timestamp);

    std::hash<std::string> hasher;
    std::uint32_t hash_value = hasher(combined);

    return hash_value;
}

unsigned long long StampedPointCloud::getTimestamp() {
    return this->timestamp;
}

pcl::PointCloud<pcl::PointXYZRGBL>::Ptr StampedPointCloud::getPointCloud() const {
    return this->cloud;
}

std::string StampedPointCloud::getOriginTopic() {
    return this->originTopic;
}

std::uint32_t StampedPointCloud::getLabel() {
    return this->label;
}

bool StampedPointCloud::isIcpTransformComputed() {
    return icpTransformComputed;
}

void StampedPointCloud::setTimestamp(unsigned long long t) {
    this->timestamp = t;
}

void StampedPointCloud::setPointCloud(pcl::PointCloud<pcl::PointXYZRGBL>::Ptr c, bool assignGeneratedLabel) {
    this->cloudSet = true;
    this->cloud = c;

    if(assignGeneratedLabel)
        this->assignLabelToPointCloud(this->cloud, this->label);
}

void StampedPointCloud::assignLabelToPointCloud(pcl::PointCloud<pcl::PointXYZRGBL>::Ptr cloud, std::uint32_t label) {
    hipError_t err = hipSuccess;
    hipStream_t stream;

    // declare the device input point array
    pcl::PointXYZRGBL *d_cloud;

    // create a stream
    if((err = hipStreamCreate(&stream)) != hipSuccess) {
        std::cerr << "Error creating the CUDA stream: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // allocate memory on the device to store the input pointcloud
    if((err = hipMalloc(&d_cloud, cloud->size() * sizeof(pcl::PointXYZRGBL))) != hipSuccess) {
        std::cerr << "Error allocating memory for the pointcloud: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // copy the input pointcloud to the device
    if((err = hipMemcpy(d_cloud, cloud->points.data(), cloud->size() * sizeof(pcl::PointXYZRGBL), hipMemcpyHostToDevice)) != hipSuccess) {
        std::cerr << "Error copying the input pointcloud to the device: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // call the kernel
    dim3 block(512);
    dim3 grid((cloud->size() + block.x - 1) / block.x);
    setPointLabelKernel<<<grid,block,0,stream>>>(d_cloud, label, cloud->size());

    // wait for the stream
    if((err = hipStreamSynchronize(stream)) != hipSuccess) {
        std::cerr << "Error waiting for the stream: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // copy the output pointcloud back to the host
    if((err = hipMemcpy(cloud->points.data(), d_cloud, cloud->size() * sizeof(pcl::PointXYZRGBL), hipMemcpyDeviceToHost)) != hipSuccess) {
        std::cerr << "Error copying the output pointcloud to the host: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // destroy the stream
    if((err = hipStreamDestroy(stream)) != hipSuccess) {
        std::cerr << "Error destroying the CUDA stream: " << hipGetErrorString(err) << std::endl;
        return;
    }
}

void StampedPointCloud::setOriginTopic(std::string origin) {
    this->originTopic = origin;
}

bool StampedPointCloud::isTransformComputed() const {
    return this->transformComputed;
}

void StampedPointCloud::applyTransform(Eigen::Affine3d tf) {
    // TODO: transform the pointcloud. have in mind they are smart pointers, 
    // attention to performance issues
    if(this->cloudSet) {
        hipError_t err = hipSuccess;
        hipStream_t stream;

        if((err = hipStreamCreate(&stream)) != hipSuccess) {
            std::cerr << "Error creating pointcloud transform stream: " << hipGetErrorString(err) << std::endl;
            return;
        }

        // allocate device memory for the pointcloud
        pcl::PointXYZRGBL *d_cloud;
        if((err = hipMalloc(&d_cloud, this->cloud->size() * sizeof(pcl::PointXYZRGBL))) != hipSuccess) {
            std::cerr << "Error allocating memory for the pointcloud: " << hipGetErrorString(err) << std::endl;
            return;
        }

        // copy the pointcloud to the device
        if((err = hipMemcpy(d_cloud, this->cloud->points.data(), this->cloud->size() * sizeof(pcl::PointXYZRGBL), hipMemcpyHostToDevice)) != hipSuccess) {
            std::cerr << "Error copying the input pointcloud to the device: " << hipGetErrorString(err) << std::endl;
            return;
        }

        // call the kernel
        dim3 block(512);
        dim3 grid((this->cloud->size() + block.x - 1) / block.x);
        transformPointKernel<<<grid,block,0,stream>>>(d_cloud, tf.matrix(), this->cloud->size());

        // wait for the stream
        if((err = hipStreamSynchronize(stream)) != hipSuccess) {
            std::cerr << "Error waiting for the stream: " << hipGetErrorString(err) << std::endl;
            return;
        }

        // copy the output pointcloud back to the host
        if((err = hipMemcpy(this->cloud->points.data(), d_cloud, this->cloud->size() * sizeof(pcl::PointXYZRGBL), hipMemcpyDeviceToHost)) != hipSuccess) {
            std::cerr << "Error copying the output pointcloud to the host: " << hipGetErrorString(err) << std::endl;
            return;
        }

        // destroy the stream
        if((err = hipStreamDestroy(stream)) != hipSuccess) {
            std::cerr << "Error destroying the CUDA stream: " << hipGetErrorString(err) << std::endl;
            return;
        }

        // pcl::transformPointCloud(*this->cloud, *this->cloud, tf);
        this->transformComputed = true;
    }
}

void StampedPointCloud::applyIcpTransform(Eigen::Matrix4f tf) {

    if(!icpTransformComputed) {

        Eigen::Matrix4d mat4d = tf.cast<double>();
        Eigen::Affine3d affine(mat4d);

        this->applyTransform(affine);

        this->icpTransformComputed = true;
    }
}

void StampedPointCloud::removePointsWithLabel(std::uint32_t label) {

    for(auto it = this->cloud->begin(); it != this->cloud->end(); it++) {
        if(it->label == label) {
            this->cloud->erase(it);
        }
    }
}
