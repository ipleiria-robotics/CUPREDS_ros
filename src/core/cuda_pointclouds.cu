#include "hip/hip_runtime.h"
#include "cuda_pointclouds.hu"

__host__ void setPointCloudLabelCuda(pcl::PointCloud<pcl::PointXYZRGBL>::Ptr cloud, std::uint32_t label) {
    hipError_t err = hipSuccess;
    hipStream_t stream;

    // declare the device input point array
    pcl::PointXYZRGBL *d_cloud;

    // create a stream
    if((err = hipStreamCreate(&stream)) != hipSuccess) {
        std::cerr << "Error creating the CUDA stream: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // allocate memory on the device to store the input pointcloud
    if((err = hipMalloc(&d_cloud, cloud->size() * sizeof(pcl::PointXYZRGBL))) != hipSuccess) {
        std::cerr << "Error allocating memory for the pointcloud: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // copy the input pointcloud to the device
    if((err = hipMemcpy(d_cloud, cloud->points.data(), cloud->size() * sizeof(pcl::PointXYZRGBL), hipMemcpyHostToDevice)) != hipSuccess) {
        std::cerr << "Error copying the input pointcloud to the device: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // call the kernel
    dim3 block(512);
    dim3 grid((cloud->size() + block.x - 1) / block.x);
    setPointLabelKernel<<<grid,block,0,stream>>>(d_cloud, label, cloud->size());

    // wait for the stream
    if((err = hipStreamSynchronize(stream)) != hipSuccess) {
        std::cerr << "Error waiting for the stream: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // copy the output pointcloud back to the host
    if((err = hipMemcpy(cloud->points.data(), d_cloud, cloud->size() * sizeof(pcl::PointXYZRGBL), hipMemcpyDeviceToHost)) != hipSuccess) {
        std::cerr << "Error copying the output pointcloud to the host: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // free the memory
    if((err = hipFree(d_cloud)) != hipSuccess) {
        std::cerr << "Error freeing the pointcloud from device memory: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // destroy the stream
    if((err = hipStreamDestroy(stream)) != hipSuccess) {
        std::cerr << "Error destroying the CUDA stream: " << hipGetErrorString(err) << std::endl;
        return;
    }
}

__global__ void setPointLabelKernel(pcl::PointXYZRGBL *points, std::uint32_t label, int num_points) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_points) {
        points[idx].label = label;
    }
}

__host__ void transformPointCloudCuda(pcl::PointCloud<pcl::PointXYZRGBL>::Ptr cloud, Eigen::Affine3d tf) {

    hipError_t err = hipSuccess;
    hipStream_t stream;

    if((err = hipStreamCreate(&stream)) != hipSuccess) {
        std::cerr << "Error creating pointcloud transform stream: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // allocate device memory for the pointcloud
    pcl::PointXYZRGBL *d_cloud;
    if((err = hipMalloc(&d_cloud, cloud->size() * sizeof(pcl::PointXYZRGBL))) != hipSuccess) {
        std::cerr << "Error allocating memory for the pointcloud: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // copy the pointcloud to the device
    if((err = hipMemcpy(d_cloud, cloud->points.data(), cloud->size() * sizeof(pcl::PointXYZRGBL), hipMemcpyHostToDevice)) != hipSuccess) {
        std::cerr << "Error copying the input pointcloud to the device: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // call the kernel
    dim3 block(512);
    dim3 grid((cloud->size() + block.x - 1) / block.x);
    transformPointKernel<<<grid,block,0,stream>>>(d_cloud, tf.matrix(), cloud->size());

    // wait for the stream
    if((err = hipStreamSynchronize(stream)) != hipSuccess) {
        std::cerr << "Error waiting for the stream: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // copy the output pointcloud back to the host
    if((err = hipMemcpy(cloud->points.data(), d_cloud, cloud->size() * sizeof(pcl::PointXYZRGBL), hipMemcpyDeviceToHost)) != hipSuccess) {
        std::cerr << "Error copying the output pointcloud to the host: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // free the memory
    if((err = hipFree(d_cloud)) != hipSuccess) {
        std::cerr << "Error freeing the pointcloud from device memory: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // destroy the stream
    if((err = hipStreamDestroy(stream)) != hipSuccess) {
        std::cerr << "Error destroying the CUDA stream: " << hipGetErrorString(err) << std::endl;
        return;
    }
}

__global__ void transformPointKernel(pcl::PointXYZRGBL *points, Eigen::Matrix4d transform, int num_points) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_points) {
        Eigen::Vector4d p(points[idx].x, points[idx].y, points[idx].z, 1.0f);
        p = transform * p;
        points[idx].x = p(0);
        points[idx].y = p(1);
        points[idx].z = p(2);
    }
}